
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>

#include <chrono>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void vec_add(float *A, float *B, float *result, int N) {
   asm(
        ".reg .pred p;\n\t"
        ".reg .f32 temp_val, a_val, b_val;\n\t"
        ".reg .u64 a, b, res;\n\t"
        ".reg .u32 tx, bx, bs, ti;\n\t"
        ".reg .u64 tia;\n\t"
        "\n\t"
        "mov.u32 tx, %tid.x;\n\t"
        "mov.u32 bx, %ctaid.x;\n\t"
        "mov.u32 bs, %ntid.x;\n\t"
        "mad.lo.u32  ti, bs, bx, tx;\n\t"
        "setp.lt.u32 p, ti, %3;\n\t"
        "cvt.u64.u32 tia, ti;\n\t"
        "@!p bra end_if;\n\t"
        "{mad.lo.u64 a, 4, tia, %0; ld.global.f32 a_val, [a]; mad.lo.u64 b, 4, tia, %1; ld.global.f32 b_val, [b]; add.f32 temp_val, a_val, b_val; mad.lo.u64 res, 4, tia, %2; st.f32 [res], temp_val;}\n\t"
        "end_if:"
   :
   : "l"(A), "l"(B), "l"(result), "r"(N)
   );
}

int main(int argc, char **argv) {
    int N = 10000;

    hiprandGenerator_t rng;
    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);

    float *d_A, *d_B, *d_result;
    hipMalloc((void **)&d_A, N * sizeof(float));
    hipMalloc((void **)&d_B, N * sizeof(float));
    hipMalloc((void **)&d_result, N * sizeof(float));

    hiprandGenerateUniform(rng, d_A, N);
    hiprandGenerateUniform(rng, d_B, N);

    int grid_dim = (N + 1023) / 1024;
    vec_add <<<grid_dim, 1024>>> (d_A, d_B, d_result, N);

    float *result, *A, *B;
    result = new float[N];
    A = new float[N];
    B = new float[N];
    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(A, d_A, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(B, d_B, N * sizeof(float), hipMemcpyDeviceToHost);
    float t;
    bool success = true;
    for (int i = 0; i < N && success; ++i) {
	t = A[i] + B[i];
        if (t - result[i] > 1e-4) {
		printf("Test failed! on entry(%d)\n", i);
                printf("%f - %f\n", result[i], t);
                success = false;
	}
    }
    if (success) {
        printf("Test Successful!\n");
    }
    
    return 0;
}

